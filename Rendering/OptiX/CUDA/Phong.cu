#include "hip/hip_runtime.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "Common.h"
#include "Light.h"


#include <stdio.h>
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );

using namespace optix;



rtDeclareVariable(rtObject, top_object, , );
rtBuffer<vtkopt::Light> lights;

rtDeclareVariable( RadiancePRD, prd,   rtPayload, );
rtDeclareVariable( optix::Ray,  ray,   rtCurrentRay, );
rtDeclareVariable( float,       t_hit, rtIntersectionDistance, );

rtDeclareVariable( float3, Ks, , );
rtDeclareVariable( float3, Kd, , );
rtDeclareVariable( float,  Ns, , );

rtDeclareVariable( IntersectionRecord, irec, attribute irec, ); 

RT_PROGRAM void LambertianClosestHit()
{
    const float3 N = faceforward( irec.N, -ray.direction, irec.Ng );
    const float3 P  = ray.origin + t_hit * ray.direction;
    //const float3 Kd = make_float3( 0.7f, 0.7f, 0.7f );

    // light loop
    float3 color = make_float3( 0.0f );
    const int num_lights = lights.size();
    for( int i =0; i < num_lights; ++i )
    {
        const vtkopt::Light light = lights[i];
        float3 L;
        float  Ldist;
        float3  Lcolor;
        if( light.type == vtkopt::Light::DIRECTIONAL )
        {
            L = -light.dir;
            Ldist = 1e8f;
            Lcolor = light.color;
        }
        else
        {
            Ldist = optix::length( light.pos - P );
            L = ( light.pos-P ) / Ldist;
            Lcolor = light.color/(Ldist*Ldist);
        }

        const float N_dot_L = optix::dot( L, N );

        float3 light_attenuation = make_float3( 0.0f );
        if( N_dot_L > 0.0f )
        {
            //
            // Calculation occlusion
            //
            OcclusionPRD shadow_prd;
            shadow_prd.occlusion = make_float3( 1.0f );
            optix::Ray shadow_ray = optix::make_Ray( P, L, OCCLUSION_RAY_TYPE, 0.001f, Ldist );
            rtTrace( top_object, shadow_ray, shadow_prd );
            light_attenuation = shadow_prd.occlusion;

            //
            // Calculate local lighting 
            //
            if( fmaxf(light_attenuation) > 0.0f )
            {
                //const float3 H = optix::normalize( L - ray.direction );
                //const float  N_dot_H = optix::dot( N, H );
                const float3 R = optix::reflect( ray.direction, N );
                const float  L_dot_R = fmaxf( optix::dot( L, R ), 0.0f );;
                color += ( Kd*N_dot_L + Ks*powf( L_dot_R, Ns ) ) * Lcolor * light_attenuation;
            }
        }


    }
    prd.result = color; 
    
}


rtDeclareVariable( OcclusionPRD, shadow_prd, rtPayload, );
RT_PROGRAM void LambertianAnyHit()
{
    shadow_prd.occlusion = make_float3( 0.0f );

}
