#include "hip/hip_runtime.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "Common.h"
#include "Light.h"


#include <stdio.h>
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );

using namespace optix;



rtDeclareVariable(rtObject, top_object, , );
rtBuffer<vtkopt::Light> lights;

rtDeclareVariable( RadiancePRD, prd,   rtPayload, );
rtDeclareVariable( optix::Ray,  ray,   rtCurrentRay, );
rtDeclareVariable( float,       t_hit, rtIntersectionDistance, );

rtDeclareVariable( IntersectionRecord, irec, attribute irec, ); 

RT_PROGRAM void LambertianClosestHit()
{
    const float3 N = faceforward( irec.N, -ray.direction, irec.Ng );
    const float3 P  = ray.origin + t_hit * ray.direction;
    const float3 Kd = make_float3( 0.7f, 0.7f, 0.7f );

    // light loop
    float3 Lc = make_float3( 0.0f );
    const int num_lights = lights.size();
    for( int i =0; i < num_lights; ++i )
    {
        const vtkopt::Light light = lights[i];
        float3 L;
        float  Ldist;
        float3  Lcolor;
        if( light.type == vtkopt::Light::DIRECTIONAL )
        {
            L = -light.dir;
            Ldist = 1e8f;
            Lcolor = light.color;
        }
        else
        {
            Ldist = optix::length( light.pos - P );
            L = ( light.pos-P ) / Ldist;
            Lcolor = light.color/(Ldist*Ldist);
        }

        const float N_dot_L = optix::dot( L, N );

        //
        // Calculation occlusion
        //
        float3 light_attenuation = make_float3(static_cast<float>( N_dot_L > 0.0f ));
        if( N_dot_L > 0.0f )
        {
            OcclusionPRD shadow_prd;
            shadow_prd.occlusion = make_float3( 1.0f );
            optix::Ray shadow_ray = optix::make_Ray( P, L, OCCLUSION_RAY_TYPE, 0.001f, Ldist );
            rtTrace( top_object, shadow_ray, shadow_prd );
            light_attenuation = shadow_prd.occlusion;
        }

        //
        // Calculate local lighting 
        //
        if( fmaxf(light_attenuation) > 0.0f )
        {
            Lc += fmaxf( 0.0f, N_dot_L ) * Lcolor * light_attenuation;
        }

    }
    prd.result = Lc*Kd; 
    
}


rtDeclareVariable( OcclusionPRD, shadow_prd, rtPayload, );
RT_PROGRAM void LambertianAnyHit()
{
    shadow_prd.occlusion = make_float3( 0.0f );

}
