#include "hip/hip_runtime.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtBuffer<uchar4, 2>   frame_buffer;
rtBuffer<float, 2>    depth_buffer;

RT_PROGRAM void draw_color()
{
    frame_buffer[launch_index] = make_uchar4(0, 255, 0, 255 );
    depth_buffer[launch_index] = 0.0f; 
}
