#include "hip/hip_runtime.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "Common.h"

using namespace optix;


rtBuffer<float3>    vertices;
rtBuffer<int3>      triangles;

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

rtDeclareVariable( IntersectionRecord, irec, attribute irec, ); 


RT_PROGRAM void TriangleMeshIntersection( int prim_idx )
{
    const int3 v_idx = triangles[prim_idx];

    const float3 p0 = vertices[ v_idx.x ];
    const float3 p1 = vertices[ v_idx.y ];
    const float3 p2 = vertices[ v_idx.z ];

    // Intersect ray with triangle
    float3 n;
    float  t, beta, gamma;
    if( intersect_triangle( ray, p0, p1, p2, n, t, beta, gamma ) )
    {
        if(  rtPotentialIntersection( t ) )
        {
            irec.Ng = normalize( n );
            rtReportIntersection( 0 );
        }
    }
}


RT_PROGRAM void TriangleMeshBoundingBox( int prim_idx, float result[6] )
{
    const int3 v_idx = triangles[prim_idx];

    const float3 v0   = vertices[ v_idx.x ];
    const float3 v1   = vertices[ v_idx.y ];
    const float3 v2   = vertices[ v_idx.z ];
    const float  area = length(cross(v1-v0, v2-v0));
  
    optix::Aabb* aabb = (optix::Aabb*)result;
  
    if( area > 0.0f && !isinf(area) )
    {
        aabb->m_min = fminf( fminf( v0, v1), v2 );
        aabb->m_max = fmaxf( fmaxf( v0, v1), v2 );
    }
    else
    {
        aabb->invalidate();
    }
}
