#include "hip/hip_runtime.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "Common.h"

using namespace optix;


rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );

rtBuffer<uchar4, 2>   frame_buffer;
rtBuffer<float, 2>    depth_buffer;

rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(float3,        pos, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );

RT_PROGRAM void PerspectiveCameraRayGen()
{
  
    const float2 d = make_float2(launch_index) / make_float2(launch_dim) * 2.f - 1.f;
    const float3 ray_origin    = pos;
    const float3 ray_direction = normalize(d.x*U + d.y*V + W);

    optix::Ray ray = optix::make_Ray(
            ray_origin,
            ray_direction,
            RADIANCE_RAY_TYPE,
            0.001f,
            RT_DEFAULT_MAX
            );


    RadiancePRD prd;
    prd.result = make_float3( 0.0f ); //ray_direction*0.5f + make_float3( 0.5f );

    rtTrace( top_object, ray, prd );

    const float3 c = fminf( prd.result, make_float3( 1.0f ) ); 
    frame_buffer[launch_index] = make_uchar4( c.x*255.99f, c.y*255.99f, c.z*255.99f, 255 );
    depth_buffer[launch_index] = 0.0f; 
}
